#include "hip/hip_runtime.h"
#ifdef WIN32
#include <float.h>
#   define isnan(x) _isnan(x)
#   define isinf(x) (! _finite(x))
#endif
#define notanum(x) (isnan(x) || isinf(x))

/* SFILE_BEGIN */
#include "essential_matrix_5pt.h"
typedef double Matches[][3];
/* SFILE_END */

// Actual expected degree of the polynomial
const int PolynomialDegree = 10;

//=============================================================================
//           Various operators on the polynomial classes
//=============================================================================

__host__ __device__ poly4_2 poly4_1::operator * (poly4_1 p2)
   {
   poly4_1 &p1 = *this;
   poly4_2 prod;

   prod(0,0)  = p1(0)*p2(0);
   prod(0,1)  = p1(0)*p2(1);
   prod(0,2)  = p1(0)*p2(2);
   prod(0,3)  = p1(0)*p2(3);

   prod(0,1) += p1(1)*p2(0);
   prod(1,1)  = p1(1)*p2(1);
   prod(1,2)  = p1(1)*p2(2);
   prod(1,3)  = p1(1)*p2(3);

   prod(0,2) += p1(2)*p2(0);
   prod(1,2) += p1(2)*p2(1);
   prod(2,2)  = p1(2)*p2(2);
   prod(2,3)  = p1(2)*p2(3);

   prod(0,3) += p1(3)*p2(0);
   prod(1,3) += p1(3)*p2(1);
   prod(2,3) += p1(3)*p2(2);
   prod(3,3)  = p1(3)*p2(3);

   return prod;
   }

__host__ __device__ poly4_3 poly4_2::operator * (poly4_1 p2)
   {
   poly4_2 &p1 = *this;
   poly4_3 prod;

   prod(0,0,0)  = p1(0,0)*p2(0);
   prod(0,0,1)  = p1(0,0)*p2(1);
   prod(0,0,2)  = p1(0,0)*p2(2);
   prod(0,0,3)  = p1(0,0)*p2(3);

   prod(0,0,1) += p1(0,1)*p2(0);
   prod(0,1,1)  = p1(0,1)*p2(1);
   prod(0,1,2)  = p1(0,1)*p2(2);
   prod(0,1,3)  = p1(0,1)*p2(3);

   prod(0,0,2) += p1(0,2)*p2(0);
   prod(0,1,2) += p1(0,2)*p2(1);
   prod(0,2,2)  = p1(0,2)*p2(2);
   prod(0,2,3)  = p1(0,2)*p2(3);

   prod(0,0,3) += p1(0,3)*p2(0);
   prod(0,1,3) += p1(0,3)*p2(1);
   prod(0,2,3) += p1(0,3)*p2(2);
   prod(0,3,3)  = p1(0,3)*p2(3);

   prod(0,1,1) += p1(1,1)*p2(0);
   prod(1,1,1)  = p1(1,1)*p2(1);
   prod(1,1,2)  = p1(1,1)*p2(2);
   prod(1,1,3)  = p1(1,1)*p2(3);

   prod(0,1,2) += p1(1,2)*p2(0);
   prod(1,1,2) += p1(1,2)*p2(1);
   prod(1,2,2)  = p1(1,2)*p2(2);
   prod(1,2,3)  = p1(1,2)*p2(3);

   prod(0,1,3) += p1(1,3)*p2(0);
   prod(1,1,3) += p1(1,3)*p2(1);
   prod(1,2,3) += p1(1,3)*p2(2);
   prod(1,3,3)  = p1(1,3)*p2(3);

   prod(0,2,2) += p1(2,2)*p2(0);
   prod(1,2,2) += p1(2,2)*p2(1);
   prod(2,2,2)  = p1(2,2)*p2(2);
   prod(2,2,3)  = p1(2,2)*p2(3);

   prod(0,2,3) += p1(2,3)*p2(0);
   prod(1,2,3) += p1(2,3)*p2(1);
   prod(2,2,3) += p1(2,3)*p2(2);
   prod(2,3,3)  = p1(2,3)*p2(3);

   prod(0,3,3) += p1(3,3)*p2(0);
   prod(1,3,3) += p1(3,3)*p2(1);
   prod(2,3,3) += p1(3,3)*p2(2);
   prod(3,3,3)  = p1(3,3)*p2(3);

#ifdef RH_DEBUG
   printf ("In poly4_2 * poly4_1\n");
   printf ("poly4_2 = \n");
   p1.print();
   printf ("poly4_1 = \n");
   p2.print();
   printf ("poly4_2 * poly4_2 = \n");
   prod.print();
#endif

   return prod;
   }

__host__ __device__ poly4_3 poly4_3::operator * (double k)
   {
   poly4_3 &p1 = *this;
   poly4_3 prod;

   prod(0,0,0) = p1(0,0,0) * k;
   prod(0,0,1) = p1(0,0,1) * k;
   prod(0,0,2) = p1(0,0,2) * k;
   prod(0,0,3) = p1(0,0,3) * k;
   prod(0,1,1) = p1(0,1,1) * k;
   prod(0,1,2) = p1(0,1,2) * k;
   prod(0,1,3) = p1(0,1,3) * k;
   prod(0,2,2) = p1(0,2,2) * k;
   prod(0,2,3) = p1(0,2,3) * k;
   prod(0,3,3) = p1(0,3,3) * k;
   prod(1,1,1) = p1(1,1,1) * k;
   prod(1,1,2) = p1(1,1,2) * k;
   prod(1,1,3) = p1(1,1,3) * k;
   prod(1,2,2) = p1(1,2,2) * k;
   prod(1,2,3) = p1(1,2,3) * k;
   prod(1,3,3) = p1(1,3,3) * k;
   prod(2,2,2) = p1(2,2,2) * k;
   prod(2,2,3) = p1(2,2,3) * k;
   prod(2,3,3) = p1(2,3,3) * k;
   prod(3,3,3) = p1(3,3,3) * k;

   return prod;
   }

__host__ __device__ poly4_3 poly4_3::operator + (poly4_3 p2)
   {
   poly4_3 &p1 = *this;
   poly4_3 sum;

   sum(0,0,0) = p1(0,0,0) + p2(0,0,0);
   sum(0,0,1) = p1(0,0,1) + p2(0,0,1);
   sum(0,0,2) = p1(0,0,2) + p2(0,0,2);
   sum(0,0,3) = p1(0,0,3) + p2(0,0,3);
   sum(0,1,1) = p1(0,1,1) + p2(0,1,1);
   sum(0,1,2) = p1(0,1,2) + p2(0,1,2);
   sum(0,1,3) = p1(0,1,3) + p2(0,1,3);
   sum(0,2,2) = p1(0,2,2) + p2(0,2,2);
   sum(0,2,3) = p1(0,2,3) + p2(0,2,3);
   sum(0,3,3) = p1(0,3,3) + p2(0,3,3);
   sum(1,1,1) = p1(1,1,1) + p2(1,1,1);
   sum(1,1,2) = p1(1,1,2) + p2(1,1,2);
   sum(1,1,3) = p1(1,1,3) + p2(1,1,3);
   sum(1,2,2) = p1(1,2,2) + p2(1,2,2);
   sum(1,2,3) = p1(1,2,3) + p2(1,2,3);
   sum(1,3,3) = p1(1,3,3) + p2(1,3,3);
   sum(2,2,2) = p1(2,2,2) + p2(2,2,2);
   sum(2,2,3) = p1(2,2,3) + p2(2,2,3);
   sum(2,3,3) = p1(2,3,3) + p2(2,3,3);
   sum(3,3,3) = p1(3,3,3) + p2(3,3,3);

   return sum;
   }

__host__ __device__ void poly4_3::operator += (poly4_3 p2)
   {
   poly4_3 &p1 = *this;

   p1(0,0,0) += p2(0,0,0);
   p1(0,0,1) += p2(0,0,1);
   p1(0,0,2) += p2(0,0,2);
   p1(0,0,3) += p2(0,0,3);
   p1(0,1,1) += p2(0,1,1);
   p1(0,1,2) += p2(0,1,2);
   p1(0,1,3) += p2(0,1,3);
   p1(0,2,2) += p2(0,2,2);
   p1(0,2,3) += p2(0,2,3);
   p1(0,3,3) += p2(0,3,3);
   p1(1,1,1) += p2(1,1,1);
   p1(1,1,2) += p2(1,1,2);
   p1(1,1,3) += p2(1,1,3);
   p1(1,2,2) += p2(1,2,2);
   p1(1,2,3) += p2(1,2,3);
   p1(1,3,3) += p2(1,3,3);
   p1(2,2,2) += p2(2,2,2);
   p1(2,2,3) += p2(2,2,3);
   p1(2,3,3) += p2(2,3,3);
   p1(3,3,3) += p2(3,3,3);
   }

__host__ __device__ poly4_3 poly4_3::operator - (poly4_3 p2)
   {
   poly4_3 &p1 = *this;
   poly4_3 dif;

   dif(0,0,0) = p1(0,0,0) - p2(0,0,0);
   dif(0,0,1) = p1(0,0,1) - p2(0,0,1);
   dif(0,0,2) = p1(0,0,2) - p2(0,0,2);
   dif(0,0,3) = p1(0,0,3) - p2(0,0,3);
   dif(0,1,1) = p1(0,1,1) - p2(0,1,1);
   dif(0,1,2) = p1(0,1,2) - p2(0,1,2);
   dif(0,1,3) = p1(0,1,3) - p2(0,1,3);
   dif(0,2,2) = p1(0,2,2) - p2(0,2,2);
   dif(0,2,3) = p1(0,2,3) - p2(0,2,3);
   dif(0,3,3) = p1(0,3,3) - p2(0,3,3);
   dif(1,1,1) = p1(1,1,1) - p2(1,1,1);
   dif(1,1,2) = p1(1,1,2) - p2(1,1,2);
   dif(1,1,3) = p1(1,1,3) - p2(1,1,3);
   dif(1,2,2) = p1(1,2,2) - p2(1,2,2);
   dif(1,2,3) = p1(1,2,3) - p2(1,2,3);
   dif(1,3,3) = p1(1,3,3) - p2(1,3,3);
   dif(2,2,2) = p1(2,2,2) - p2(2,2,2);
   dif(2,2,3) = p1(2,2,3) - p2(2,2,3);
   dif(2,3,3) = p1(2,3,3) - p2(2,3,3);
   dif(3,3,3) = p1(3,3,3) - p2(3,3,3);

   return dif;
   }

__host__ __device__ poly4_2 poly4_2::operator + (poly4_2 p2)
   {
   poly4_2 &p1 = *this;
   poly4_2 sum;

   sum(0,0) = p1(0,0) + p2(0,0);
   sum(0,1) = p1(0,1) + p2(0,1);
   sum(0,2) = p1(0,2) + p2(0,2);
   sum(0,3) = p1(0,3) + p2(0,3);
   sum(1,1) = p1(1,1) + p2(1,1);
   sum(1,2) = p1(1,2) + p2(1,2);
   sum(1,3) = p1(1,3) + p2(1,3);
   sum(2,2) = p1(2,2) + p2(2,2);
   sum(2,3) = p1(2,3) + p2(2,3);
   sum(3,3) = p1(3,3) + p2(3,3);

   return sum;
   }

__host__ __device__ void poly4_2::operator += (poly4_2 p2)
   {
   poly4_2 &p1 = *this;

   p1(0,0) += p2(0,0);
   p1(0,1) += p2(0,1);
   p1(0,2) += p2(0,2);
   p1(0,3) += p2(0,3);
   p1(1,1) += p2(1,1);
   p1(1,2) += p2(1,2);
   p1(1,3) += p2(1,3);
   p1(2,2) += p2(2,2);
   p1(2,3) += p2(2,3);
   p1(3,3) += p2(3,3);
   }

__host__ __device__ poly4_2 poly4_2::operator - (poly4_2 p2)
   {
   poly4_2 &p1 = *this;
   poly4_2 dif;

   dif(0,0) = p1(0,0) - p2(0,0);
   dif(0,1) = p1(0,1) - p2(0,1);
   dif(0,2) = p1(0,2) - p2(0,2);
   dif(0,3) = p1(0,3) - p2(0,3);
   dif(1,1) = p1(1,1) - p2(1,1);
   dif(1,2) = p1(1,2) - p2(1,2);
   dif(1,3) = p1(1,3) - p2(1,3);
   dif(2,2) = p1(2,2) - p2(2,2);
   dif(2,3) = p1(2,3) - p2(2,3);
   dif(3,3) = p1(3,3) - p2(3,3);

   return dif;
   }

__host__ __device__ poly4_1 poly4_1::operator + (poly4_1 p2)
   {
   poly4_1 &p1 = *this;
   poly4_1 sum;

   sum(0) = p1(0) + p2(0);
   sum(1) = p1(1) + p2(1);
   sum(2) = p1(2) + p2(2);
   sum(3) = p1(3) + p2(3);

   return sum;
   }

__host__ __device__ poly4_1 poly4_1::operator - (poly4_1 p2)
   {
   poly4_1 &p1 = *this;
   poly4_1 dif;

   dif(0) = p1(0) - p2(0);
   dif(1) = p1(1) - p2(1);
   dif(2) = p1(2) - p2(2);
   dif(3) = p1(3) - p2(3);

   return dif;
   }

//=============================================================================

__host__ __device__ poly4_3 polydet4 (EmatrixSet_5pt E)
   {
   // Takes the determinant of a polynomial
   poly4_3 det = 
      (E(1,1)*E(2,2) - E(2,1)*E(1,2)) * E(0,0) +
      (E(2,1)*E(0,2) - E(0,1)*E(2,2)) * E(1,0) +
      (E(0,1)*E(1,2) - E(1,1)*E(0,2)) * E(2,0);

#ifdef RH_DEBUG
   printf ("Det =\n");
   det.print();
#endif

   return det;
   }

#define FULL_TRACE
#ifdef  FULL_TRACE
__host__ __device__ poly4_2 traceEEt (EmatrixSet_5pt E)
   {
   // Takes the trace of E E' -- returns a quadratic polynomial
   // Trace of product is the elementwise product of the elements

   poly4_2 tr = E(0,0) * E(0, 0) + E(0,1) * E(0, 1) + E(0,2) * E(0, 2) 
            + E(1,0) * E(1, 0) + E(1,1) * E(1, 1) + E(1,2) * E(1, 2) 
            + E(2,0) * E(2, 0) + E(2,1) * E(2, 1) + E(2,2) * E(2, 2);

#ifdef RH_DEBUG
   printf ("Trace is:\n");
   tr.print();
#endif

   return tr;
   }

#else

__host__ __device__ poly4_2 traceEEt (EmatrixSet_5pt E)
   {
   // We know that the trace has a simple form, provided that the
   // E-matrix basis is orthogonal.

   poly4_2 tr;  
   tr.clear();
   tr(0,0) = 1.0;
   tr(1,1) = 1.0;
   tr(2,2) = 1.0;
   tr(3,3) = 1.0;

   return tr;
   }

#endif

__host__ __device__ void mono_coeff (poly4_3 B, EquationSet A, int n)
   {
   // Extracts the monomial coefficients in x and y (with z = 1) from
   // a cubic homogeneous polynomial. Returns 4 vectors (degrees 0 to 3 in w)

   // Make some constants to make the code easier to read

   // Degrees of terms in w
   const int w0 = 0;
   const int w1 = 1;
   const int w2 = 2;
   const int w3 = 3;

   // Linear variables
   const int w = 0;
   const int x = 1;
   const int y = 2;
   const int z = 3;

   // Monomials
   const int xx  = 3;
   const int xy  = 4;
   const int yy  = 5;
   const int xxx = 6;
   const int xxy = 7;
   const int xyy = 8;
   const int yyy = 9;

   // Terms in w^0
   A[w0][n][ 0  ] = B(z, z, z);
   A[w0][n][ x  ] = B(x, z, z);
   A[w0][n][ y  ] = B(y, z, z);
   A[w0][n][ xx ] = B(x, x, z);
   A[w0][n][ yy ] = B(y, y, z);
   A[w0][n][ xy ] = B(x, y, z);
   A[w0][n][ xxx] = B(x, x, x);
   A[w0][n][ xxy] = B(x, x, y);
   A[w0][n][ xyy] = B(x, y, y);
   A[w0][n][ yyy] = B(y, y, y);

   // Terms in w^1
   A[w1][n][ 0  ] = B(w, z, z);
   A[w1][n][ x  ] = B(w, x, z);
   A[w1][n][ y  ] = B(w, y, z);
   A[w1][n][ xx ] = B(w, x, x);
   A[w1][n][ yy ] = B(w, y, y);
   A[w1][n][ xy ] = B(w, x, y);

   // Terms in w^2
   A[w2][n][ 0  ] = B(w, w, z);
   A[w2][n][ x  ] = B(w, w, x);
   A[w2][n][ y  ] = B(w, w, y);

   // Terms in w^3
   A[w3][n][ 0  ] = B(w, w, w);
   }

__host__ __device__ void EEeqns_5pt (EmatrixSet_5pt E, EquationSet A)
   {
   //
   // Computes the equations that will be used to input to polyeig.
   //    void EEeqns_5pt(E, A)
   // where E has dimensions E(3, 3, 4).  The output is a matrix
   // of dimension A(4, 10, 10), where A(i, :, :) is the coeffient of w^{i-1}
   //

   // Makes all the equations from the essential matrix E
   // First of all, set the equations to zero
   memset (&(A[0][0][0]), 0, sizeof(EquationSet));
   // Find the trace - this is a quadratic polynomial
   poly4_2 tr = traceEEt(E);
   // First equation is from the determinant
   mono_coeff (polydet4(E), A, 0);
   // Other equations from the equation 2 E*E'*E - tr(E*E') E = 0
   // In the following loop, we compute EE'E(i,j) = sum_pq E(i,p)*E(q,p)*E(q,j)
   // The way this is done is optimized for speed.  We compute first the matrix
   // EE'(i, q) and then use this to accumulate EE'E(i, j)

   int eqn = 1;  // Count on the next equation
   for (int i=0; i<3; i++)
      {
      // An array of cubic polynomials, one for each j = 0 ... 2
      poly4_3 EEE_i[3];  // Will hold (EE'E)(i,j)
      for (int j=0; j<3; j++) EEE_i[j].clear();

      // Compute each EE'(i,q) = sum_p E(i,p) E(q,p)
      for (int q=0; q<3; q++)
         {
         // Accumulate EE(i, q)
         poly4_2 EE_iq; EE_iq.clear();
         for (int p=0; p<3; p++)
            EE_iq += E(i,p) * E(q,p);

         // Now, accumulate EEE(ij) = sum_q  EE'(i,q) * E(q, j)
         for (int j=0; j<3; j++)
            EEE_i[j] += EE_iq * E(q,j);
         }

      // Now, EE'E(i,j) is computed for this i and all j
      // We can complete the computation of the coefficients from EE'E(i, j)
      for (int j=0; j<3; j++)
         mono_coeff(EEE_i[j]*2.0 - tr* E(i,j), A, eqn++);
      }
   }

__host__ __device__ void null_space_solve_3x3_half_pivot (double A[3][3], double &x, double &y)
   {
   //
   // Solve for the null-space of the matrix.  The value returned is
   // (x, y), where (1, x, y) is the generator of the null-space.
   //

   // This time we will do pivoting
   int p1;
   double f0 = fabs(A[0][2]), f1 = fabs(A[1][2]), f2 = fabs(A[2][2]);
   if (f0 > f1) p1 = (f0>f2)? 0 : 2;
   else p1 = (f1>f2) ? 1 : 2;

   // The other two rows
   int r1 = (p1+1)%3, r2 = (p1+2)%3;

   // Now, use this to pivot
   double fac = A[r1][2] / A[p1][2];
   A[r1][0] -= fac * A[p1][0];
   A[r1][1] -= fac * A[p1][1];

   fac = A[r2][2] / A[p1][2];
   A[r2][0] -= fac * A[p1][0];
   A[r2][1] -= fac * A[p1][1];
  
   // Second pivot - largest element in column 1
   int p2 = fabs(A[r1][1]) > fabs(A[r2][1]) ? r1 : r2;
   
   // Now, read off the values - back substitution
   x = - A[p2][0]               / A[p2][1];
   y = -(A[p1][0] + A[p1][1]*x) / A[p1][2];
   }

#if 0
//
// THIS DOES NOT COMPILE BECAUSE OF THE USE OF rhMatrix.
// IT IS POSSIBLE THAT IT SHOULD BE USED>
//
__host__ __device__ void null_space_solve_3x3 (double A[3][3], double &x, double &y)
   {
   // Solve for the null-space of the matrix
   rhMatrix AA (3, 3, &(A[0][0]));
   rhVector D(3);
   rhMatrix V(3, 3);

   // Take the SVD
   svd(AA, D, V);

   if (V[0][2] == 0.0)
      {
      // Just something that stops it from crashing
      x = 0.0;
      y = 0.0;
      }
   else
      {
      x = V[1][2] / V[0][2];
      y = V[2][2] / V[0][2];
      }
   }
#endif

__host__ __device__ void null_space_solve_5x9_nopivot (double A[5][9], EmatrixSet_5pt &E)
   {
   // This will compute the set of solutions for the equations
   // Sweep out one column at a time, starting with highest column number

   // We do Gaussian elimination to convert M to the form M = [X | I]
   // Then the null space will be [-I | X].

   // For present, this is done without pivoting.  
   // Mostly, do not need to actually change right hand part (that becomes I)

   const int lastrow  = 4;
   const int firstcol = 4; // First column to do elimination to make I
   const int lastcol  = 8; 

   // First sweep is to get rid of the above diagonal parts
   for (int col=lastcol; col>firstcol; col--)  // No need to do first col
      {
      // Remove column col
      const int row = col-firstcol; // Row to pivot around
      const double pivot = A[row][col];

      // Sweep out all rows up to the current one 
      for (int i=0; i<row; i++)
         {
         // This factor of the pivot row is to subtract from row i
         const double fac = A[i][col] / pivot;

         // Constant terms
         for (int j=0; j<col; j++)
            A[i][j] -= fac * A[row][j];
         }
      }

   // Now, do backward sweep to clear below the diagonal
   for (int col=firstcol; col<lastcol; col++) // No need to do lastcol
      {
      // Remove column col
      const int row = col-firstcol; // Row to pivot around
      const double pivot = A[row][col];

      // Sweep out all rows up to the current one 
      for (int i=row+1; i<=lastrow; i++)
         {
         // This factor of the pivot row is to subtract from row i
         const double fac = A[i][col] / pivot;

         // Constant terms
         for (int j=0; j<firstcol; j++)
            A[i][j] -= fac * A[row][j];
         }
      }

   // Make this into a matrix of solutions
   double fac;
   E(0, 0) = poly4_1(1.0, 0.0, 0.0, 0.0);
   E(0, 1) = poly4_1(0.0, 1.0, 0.0, 0.0);
   E(0, 2) = poly4_1(0.0, 0.0, 1.0, 0.0);
   E(1, 0) = poly4_1(0.0, 0.0, 0.0, 1.0);
   fac = -1.0/A[0][4];
   E(1, 1) = poly4_1(fac*A[0][0], fac*A[0][1], fac*A[0][2], fac*A[0][3]);
   fac = -1.0/A[1][5];
   E(1, 2) = poly4_1(fac*A[1][0], fac*A[1][1], fac*A[1][2], fac*A[1][3]);
   fac = -1.0/A[2][6];
   E(2, 0) = poly4_1(fac*A[2][0], fac*A[2][1], fac*A[2][2], fac*A[2][3]);
   fac = -1.0/A[3][7];
   E(2, 1) = poly4_1(fac*A[3][0], fac*A[3][1], fac*A[3][2], fac*A[3][3]);
   fac = -1.0/A[4][8];
   E(2, 2) = poly4_1(fac*A[4][0], fac*A[4][1], fac*A[4][2], fac*A[4][3]);

// #define USE_TEST_VALUES
#ifdef  USE_TEST_VALUES

   // Put an artificial value in 
   E(0,0)(0) =  2; E(0,1)(0) =   4; E(0,2)(0) = -1;
   E(1,0)(0) =  4; E(1,1)(0) =   5; E(1,2)(0) = -8;
   E(2,0)(0) =  2; E(2,1)(0) = -11; E(2,2)(0) =  8;

   E(0,0)(1) =  0; E(0,1)(1) =  -1; E(0,2)(1) =  2;
   E(1,0)(1) =  1; E(1,1)(1) =   7; E(1,2)(1) =  1;
   E(2,0)(1) = -2; E(2,1)(1) =   6; E(2,2)(1) =  7;

   E(0,0)(2) =  2; E(0,1)(2) =  -3; E(0,2)(2) =  7;
   E(1,0)(2) =  1; E(1,1)(2) =  -3; E(1,2)(2) = -9;
   E(2,0)(2) =  4; E(2,1)(2) =   1; E(2,2)(2) = -9;

   E(0,0)(3) =  5; E(0,1)(3) =   2; E(0,2)(3) =  7;
   E(1,0)(3) =  1; E(1,1)(3) =  -2; E(1,2)(3) = -4;
   E(2,0)(3) =  5; E(2,1)(3) =  -1; E(2,2)(3) =  8;

#endif
   }

__host__ __device__ void null_space_solve_5x9 (double A[9][9], EmatrixSet_5pt &E)
   {
   // This will compute the set of solutions for the equations
   // We do orthogonal reduction of the rows of the matrix
   const int nrows = 9;
   const int ncols = 9;

   // First, fill out some random entries in the remaining rows
   const double PPi = 3.18730379;       // Basically a random number
   double ran = PPi;
   for (int i=5; i<nrows; i++)
      {
      for (int j=0; j<ncols; j++)
         {
         ran *= PPi;
         ran = 2.0 * (ran - floor(ran)) - 1.0;
         A[i][j] = ran;
         }
      }
   // Now, do Gram-Schmidt
   for (int row=0; row<nrows; row++)
      {
      // Normalize the row
      double sum = 0.0;
      for (int j=0; j<ncols; j++) sum += A[row][j]*A[row][j];
      double fac = 1.0 / sqrt(sum);
      for (int j=0; j<ncols; j++) A[row][j] *= fac;

      // Use to sweep out the subsequent rows
      for (int i=row+1; i<nrows; i++)
         {
         // Inner product of row i and row j
         double prod = 0.0;
         for (int j=0; j<ncols; j++)
            prod += A[row][j]*A[i][j]; // Inner product
         for (int j=0; j<ncols; j++) A[i][j] -= prod * A[row][j];
         }
      }

   // Make this into a matrix of solutions
   int count = 0;
   for (int i=0; i<3; i++)
      for (int j=0; j<3; j++)
         {
         E(i,j) = poly4_1(A[5][count], A[6][count], A[7][count], A[8][count]);
         count++;
         }
   }

__host__ __device__ void Ematrix_5pt(Matches q, Matches qp, EmatrixSet_5pt &E, EquationSet &A)
   {
   // Computes the E-matrix from match inputs
   // A matrix to solve linearly for the ematrix
   double M[9][9];
   memset (&(M[0][0]), 0, sizeof (M));

   for (int i=0; i<5; i++)
      {
      M[i][0] = qp[i][0]*q[i][0];
      M[i][1] = qp[i][0]*q[i][1]; 
      M[i][2] = qp[i][0]*q[i][2];
      M[i][3] = qp[i][1]*q[i][0];
      M[i][4] = qp[i][1]*q[i][1]; 
      M[i][5] = qp[i][1]*q[i][2]; 
      M[i][6] = qp[i][2]*q[i][0];
      M[i][7] = qp[i][2]*q[i][1];
      M[i][8] = qp[i][2]*q[i][2]; 
      }
   // Solve using null_space_solve
   null_space_solve_5x9 (M, E);

   // Now, get the equations
   EEeqns_5pt(E, A);
   }

__host__ __device__ void sweep_up (EquationSet A, int row, int col, int degree)
   {
   // Use the given pivot point to sweep out above the pivot
   const int num1 = 6; // number of nonzero columns of A in degree 1
   const int num2 = 3; // number of nonzero columns of A in degree 2
   const int num3 = 1; // number of nonzero columns of A in degree 3

   // Find the pivot value
   const double pivot = A[degree][row][col];

   // Sweep out all rows up to the current one 
   for (int i=0; i<row; i++)
      {
      // This factor of the pivot row is to subtract from row i
      const double fac = A[degree][i][col] / pivot;

      // Constant terms
      for (int j=0; j<=col; j++)
         A[0][i][j] -= fac * A[0][row][j];

      // Degree 1 terms
      for (int j=0; j<num1; j++)
         A[1][i][j] -= fac * A[1][row][j];

      // Degree 2 terms
      for (int j=0; j<num2; j++)
         A[2][i][j] -= fac * A[2][row][j];

      // Degree 3 terms
      for (int j=0; j<num3; j++)
         A[3][i][j] -= fac * A[3][row][j];
      }
   }

__host__ __device__ void sweep_down (EquationSet A, int row, int col, int degree, int lastrow)
   {
   // Use the given pivot point to sweep out below the pivot
   const int num1 = 6; // number of nonzero columns of A in degree 1
   const int num2 = 3; // number of nonzero columns of A in degree 2
   const int num3 = 1; // number of nonzero columns of A in degree 3

   // The value of the pivot point
   const double pivot = A[degree][row][col];

   // Sweep out all rows up to the current one 
   for (int i=row+1; i<=lastrow; i++)
      {
      // This factor of the pivot row is to subtract from row i
      const double fac = A[degree][i][col] / pivot;

      // Constant terms
      for (int j=0; j<=col; j++)
         A[0][i][j] -= fac * A[0][row][j];

      // Degree 1 terms
      for (int j=0; j<num1; j++)
         A[1][i][j] -= fac * A[1][row][j];

      // Degree 2 terms
      for (int j=0; j<num2; j++)
         A[2][i][j] -= fac * A[2][row][j];

      // Degree 3 terms
      for (int j=0; j<num3; j++)
         A[3][i][j] -= fac * A[3][row][j];
      }
   }

__host__ __device__ inline void swap (double &a, double &b)
   {
   double temp = a; a = b; b = temp;
   }

__host__ __device__ void pivot (EquationSet A, int last, int deg, int dummy)
   {
   // Pivot so that the largest element in the column is in the diagonal

   // Use the given pivot point to sweep out below the pivot
   const int num1 = 6; // number of nonzero columns of A in degree 1
   const int num2 = 3; // number of nonzero columns of A in degree 2
   const int num3 = 1; // number of nonzero columns of A in degree 3

   // Find the maximum value in the column
   double maxval = fabs(A[deg][last][last]);
   int row = last;
   for (int i=0; i<last; i++)
      {
      if (fabs(A[deg][i][last]) > maxval)
         {
         row = i;
         maxval = fabs(A[deg][i][last]);
         }
      }

   // If is in the maximum position, then return.
   if (row == last) return;

   // Otherwise, swap
   // Constant terms
   for (int j=0; j<=last; j++)
      swap(A[0][last][j], A[0][row][j]);

   // Degree 1 terms
   for (int j=0; j<num1; j++)
      swap(A[1][last][j], A[1][row][j]);

   // Degree 2 terms
   for (int j=0; j<num2; j++)
      swap(A[2][last][j], A[2][row][j]);

   // Degree 3 terms
   for (int j=0; j<num3; j++)
      swap(A[3][last][j], A[3][row][j]);
   }

__host__ __device__ void reduce_Ematrix (EquationSet A)
   {
   // This reduces the equation set to 3 x 3.  In this version there is
   // no pivoting, which relies on the pivots to be non-zero.

   // Relies on the particular form of the A matrix to reduce it
   // That means that there are several rows of zero elements in different
   // degrees, as given below.

   // Sweeping out the constant terms to reduce to 6 x 6
   pivot (A, 9, 0, 8); sweep_up (A, 9, 9, 0);
   pivot (A, 8, 0, 7); sweep_up (A, 8, 8, 0);
   pivot (A, 7, 0, 6); sweep_up (A, 7, 7, 0);
   pivot (A, 6, 0, 5); sweep_up (A, 6, 6, 0);

   // Now, the matrix is 6 x 6.  Next we need to handle linear terms
   pivot (A, 5, 0, 4); sweep_up (A, 5, 5, 0);
   pivot (A, 4, 0, 3); sweep_up (A, 4, 4, 0);
   pivot (A, 3, 0, 2); sweep_up (A, 3, 3, 0);

   int lastrow = 5;
   sweep_down (A, 3, 3, 0, lastrow);
   sweep_down (A, 4, 4, 0, lastrow);

   // Also sweep out the first-order terms
   sweep_up   (A, 2, 5, 1);
   sweep_up   (A, 1, 4, 1);

   sweep_down (A, 0, 3, 1, lastrow);
   sweep_down (A, 1, 4, 1, lastrow);
   sweep_down (A, 2, 5, 1, lastrow);

   // Now, sweep out the x terms by increasing the degree
   for (int i=0; i<3; i++)
      {
      double fac = A[1][i][3+i] / A[0][3+i][3+i];

      // Introduces 4-th degree term
      A[4][i][0] = -A[3][i+3][0] * fac;

      // Transfer terms of degree 0 to 3
      for (int j=0; j<3; j++)
         {
         A[3][i][j] -= A[2][i+3][j] * fac;
         A[2][i][j] -= A[1][i+3][j] * fac;
         A[1][i][j] -= A[0][i+3][j] * fac;
         }
      }
   }


__host__ __device__ inline void one_cofactor (EquationSet A, Polynomial poly, 
   int r0, int r1, int r2)
   {
   // Computes one term of the 3x3 cofactor expansion

   // Get a polynomial to hold a 2x2 determinant
   double two[7];
   memset (&(two[0]), 0, 7*sizeof(double));

   // Compute the 2x2 determinant - results in a 6-degree polynomial
   for (int i=0; i<=3; i++)
      for (int j=0; j<=3; j++)
         two [i+j] += A[i][r1][1]*A[j][r2][2] - A[i][r2][1]*A[j][r1][2];

   // Now, multiply by degree 4 polynomial
   for (int i=0; i<=6; i++)
      for (int j=0; j<=4; j++)
         poly [i+j] += A[j][r0][0]*two[i];
   }

__host__ __device__ void compute_determinant (EquationSet A, Polynomial poly)
   {
   // Does the final determinant computation to return the determinant
   // Input is a 3x3 matrix of polynomialsm A,
   // Output is poly (degree 10)

   // Clear out the polynomial
   memset (&(poly[0]), 0, (PolynomialDegree+1)*sizeof(double));

   // Now, the three cofactors
   one_cofactor (A, poly, 0, 1, 2);
   one_cofactor (A, poly, 1, 2, 0);
   one_cofactor (A, poly, 2, 0, 1);
   }


// Declaration of the function to find roots
__host__ __device__ int find_real_roots_sturm( 
   double *p, int order, double *roots, int *nroots, bool non_neg = false);

__host__ __device__ void compute_E_matrix (EmatrixSet_5pt &Es, EquationSet &A, double w, Ematrix &E)
   {
   // Compute the essential matrix corresponding to this root
   double w2 = w*w;
   double w3 = w2*w;
   double w4 = w3*w;
 
   // Form equations to solve
   double M[3][3];
   for (int i=0; i<3; i++)
      {
      for (int j=0; j<3; j++)
         {
         M[i][j] = A[0][i][j] + w*A[1][i][j] + w2*A[2][i][j] + w3*A[3][i][j];
         }

      // Only the first row has degree 4 terms
      M[i][0] += w4*A[4][i][0];
      }

   // Now, find the solution
   double x, y;
   null_space_solve_3x3_half_pivot (M, x, y);

   //-----------------------------------------------------
   // Insurance that it worked
   // If this failed, then try again with different method

#if 0
   //
   // POSSIBLY THIS SHOULD BE USED, BUT IT DOES NOT COMPILE.
   //
   if (notanum(x) || notanum(y))
      {
      // Do it again
      for (int i=0; i<3; i++)
         {
         for (int j=0; j<3; j++)
            {
            M[i][j] = A[0][i][j] + w*A[1][i][j] + w2*A[2][i][j] + w3*A[3][i][j];
            }

         // Only the first row has degree 4 terms
         M[i][0] += w4*A[4][i][0];
         }

      // Solve using safer SVD solver
      null_space_solve_3x3 (M, x, y);
      }
#endif

   //-----------------------------------------------------

   // Multiply out the solution to get the essential matrix
   for (int i=0; i<3; i++)
      for (int j=0; j<3; j++)
         {
         poly4_1 &p = Es(i, j);
         E[i][j] = w*p(0) + x*p(1) + y*p(2) + p(3);
         }
   }


__host__ __device__ void compute_E_matrices_optimized (
     Matches q, Matches qp,
     Ematrix Ematrices[10],
     int &nroots
     )
   {
   // Get the matrix set
   EquationSet A;
   EmatrixSet_5pt E;
   Ematrix_5pt(q, qp, E, A);

   // Now, reduce its dimension to 3 x 3
   reduce_Ematrix(A);

   // Finally, get the 10-th degree polynomial out of this
   Polynomial poly;
   compute_determinant(A, poly);

   // Find the roots
   double roots[PolynomialDegree];
   find_real_roots_sturm(poly, PolynomialDegree, roots, &nroots); // ~50MB

   // Now, get the ematrices
   for (int i=0; i<nroots; i++)
      compute_E_matrix(E, A, roots[i], Ematrices[i]);
   }

